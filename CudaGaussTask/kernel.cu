#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include "Common.h"

__global__ void forwardPropagation(float *, float *, int); 

void calculateTriangleMatrix(float *temp_h , int numvar , float *temp1_h) 
{ 
    float *a_d , *b_d; 
    
    //Memory allocation on the device 
    hipMalloc(&a_d,sizeof(float)*(numvar)*(numvar+1)); 
    hipMalloc(&b_d,sizeof(float)*(numvar)*(numvar+1)); 
    
    //Copying data to device from host 
    hipMemcpy(a_d, temp_h, sizeof(float)*numvar*(numvar+1),hipMemcpyHostToDevice); 
    
    //Defining size of Thread Block 
    dim3 dimBlock(numvar + 1, numvar, 1); 
    dim3 dimGrid(1,1,1); 
    
    //Kernel call 
    forwardPropagation<<<dimGrid , dimBlock>>>(a_d, b_d, numvar); 
    
    //Coping data to host from device 
    hipMemcpy(temp1_h,b_d, sizeof(float)*numvar*(numvar+1), hipMemcpyDeviceToHost);
    hipFree(a_d); 
    hipFree(b_d);
}

__global__ void forwardPropagation(float *a_d , float *b_d ,int size) 
{
    int idx = threadIdx.x ; 
    int idy = threadIdx.y ; 
    
    //Allocating memory in the share memory of the device 
    __shared__ float temp[16][16]; 
    
    //Copying the data to the shared memory 
    temp[idy][idx] = a_d[(idy * (size+1)) + idx] ; 
    
    for(int i =1 ; i<size ;i++) 
    { 
        if((idy + i) < size) // NO Thread divergence here 
        { 
            float var1 =(-1)*( temp[i-1][i-1]/temp[i+idy][i-1]); 
            temp[i+idy][idx] = temp[i-1][idx] +((var1) * (temp[i+idy][idx]));
        } 
        __syncthreads(); //Synchronizing all threads before Next iterat ion 
    } 
    
    b_d[idy*(size+1) + idx] = temp[idy][idx]; 
}
//����������� ����������� �������� �� ������

__host__ void backPropagation(float **result, float *b_h, int numvar) {
	float sum, rvalue; 
	int j;
	*result = (float*)malloc(sizeof(float)*(numvar)); 
    
	for(int i = 0; i < numvar;i++) 
    { 
        (*result)[i] = 1.0;
    }
	//�������� ���
    for(int i=numvar-1 ; i>=0 ; i--) 
    { 
        sum = 0.0 ;
		#pragma omp parallel for reduction(+:sum) private(j, numvar) shared(result, b_h)
        for(j = numvar-1; j > i ;j--) 
        { 
            sum = sum + (*result)[j]*b_h[i*(numvar+1) + j]; 
        }
        rvalue = b_h[i*(numvar+1) + numvar] - sum ; 
        (*result)[i] = rvalue / b_h[i *(numvar+1) + j];
    } 
}

int main()
{	
	float *a_h = NULL;
    float *b_h = NULL;

    float *result, sum, rvalue; 
    int numvar, j;
	
	numvar = readMatrix(&a_h);	
	b_h = (float*)malloc(sizeof(float) * numvar * (numvar+1));
	result = (float*)malloc(sizeof(float)*(numvar));

	/***���������������� ������***/
	/*sequenceForwardPropagation(&a_h, numvar);

	sequenceBackPropagation(a_h, &result, numvar);
	
	for(int i = 0; i < numvar; i++) 
    { 
        printf("[X%d] = %+f\n", i , result[i]); 
    }*/

	/***������������ ������***/
	calculateTriangleMatrix(a_h , numvar , b_h); 
        
	result = (float*)malloc(sizeof(float)*(numvar)); 
    for(int i = 0; i < numvar;i++) 
    { 
        result[i] = 1.0;
    }
    backPropagation(&result, b_h, numvar);
	#pragma omp parallel for
    for(int i = 0; i < numvar; i++) 
    { 
        printf("[X%d] = %+f\n", i ,result[i]); 
    }

	system("pause");
    return 0;
}
